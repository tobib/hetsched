#include "hip/hip_runtime.h"
/* http://majuric.org/software/cudamd5/ */

// CUDA MD5 hash calculation implementation (A: mjuric@ias.edu).
//
// A very useful link: http://people.eku.edu/styere/Encrypt/JS-MD5.html
//

#define RSA_KERNEL md5_v2

#include <stdio.h>
#include "cutil.h"
#include <cutil_inline.h>
#include "worker_md5.h"

#if ORIGINAL_WORD_LENGTH > 16*4
	#error "Word length too long for CUDA implementation"
#endif

#define count_t ullong 

typedef unsigned int uint;
typedef unsigned long long ullong;

//
// On-device variable declarations
//

extern __shared__ uint memory[];	// on-chip shared memory
__constant__ uint k[64], rconst[16];	// constants (in fast on-chip constant cache)
__constant__ uint steps[ORIGINAL_WORD_LENGTH];		// calculation helper to convert a number to a word using the MD5POOL

//
// MD5 routines (straight from Wikipedia's MD5 pseudocode description)
//

__device__ inline uint leftrotate (uint x, uint c)
{
	return (x << c) | (x >> (32-c));
}

__device__ inline uint r(const uint i)
{
	return rconst[(i / 16) * 4 + i % 4];
}

// Accessor for w[16] array. Naively, this would just be w[i]; however, this
// choice leads to worst-case-scenario access pattern wrt. shared memory
// bank conflicts, as the same indices in different threads fall into the
// same bank (as the words are 16 uints long). The packing below causes the
// same indices in different threads of a warp to map to different banks. In
// testing this gave a ~40% speedup.
//
// PS: An alternative solution would be to make the w array 17 uints long
// (thus wasting a little shared memory)
//
__device__ inline uint &getw(uint *w, const int i)
{
	return w[(i+threadIdx.x) % 16];
}

__device__ inline uint getw(const uint *w, const int i)	// const- version
{
	return w[(i+threadIdx.x) % 16];
}


__device__ inline uint getk(const int i)
{
	return k[i];	// Note: this is as fast as possible (measured)
}

__device__ void step(const uint i, const uint f, const uint g, uint &a, uint &b, uint &c, uint &d, const uint *w)
{
	uint temp = d;
	d = c;
	c = b;
	b = b + leftrotate((a + f + getk(i) + getw(w, g)), r(i));
	a = temp;
}

__device__ void inline md5(const uint *w, uint &a, uint &b, uint &c, uint &d)
{
	const uint a0 = 0x67452301;
	const uint b0 = 0xEFCDAB89;
	const uint c0 = 0x98BADCFE;
	const uint d0 = 0x10325476;

	//Initialize hash value for this chunk:
	a = a0;
	b = b0;
	c = c0;
	d = d0;

	uint f, g, i = 0;
	for(; i != 16; i++)
	{
		f = (b & c) | ((~b) & d);
		g = i;
		step(i, f, g, a, b, c, d, w);
	}

	for(; i != 32; i++)
	{
		f = (d & b) | ((~d) & c);
		g = (5*i + 1) % 16;
		step(i, f, g, a, b, c, d, w);
	}

	for(; i != 48; i++)
	{
		f = b ^ c ^ d;
		g = (3*i + 5) % 16;
		step(i, f, g, a, b, c, d, w);
	}

	for(; i != 64; i++)
	{
		f = c ^ (b | (~d));
		g = (7*i) % 16;
		step(i, f, g, a, b, c, d, w);
	}

	a += a0;
	b += b0;
	c += c0;
	d += d0;
}

/* 
 * prepare a 56-byte (maximum) wide md5 message by appending the 64-bit length
 * it will be padded with 0 and will contain the messaged 'packed' into an uint array
 *
 * NOTE: This function will fail badly if it is called with a number >= steps[0] * NUM_OF_CHARS
 *
 * word is assumed to be a w[16] array and is thus accessed via getw()
 */
__device__ void number2paddedword (count_t number, uint *word)
{
	int srciter=0;
	int dstiter=0;
	char md5pool[sizeof(MD5POOL)] = MD5POOL;
  char curChar;
  int shiftoffset = 0; /* current offset to shift the next char into the uint */
  uint nextArrayUint = 0;

  /*
	 * Special case: Length of words is 0 or 1
	 * These cases can be determined at compile time and can therefore
	 * be optimized away by the compiler
	 */
	if (ORIGINAL_WORD_LENGTH < 1)
		return;
	
	/* loop through the source word */
  for (srciter = 0; srciter < ORIGINAL_WORD_LENGTH; ++srciter) {
    /* Decide if we have to encode a specific char or just md5pool[0] */
		if (number >= steps[srciter] || srciter == ORIGINAL_WORD_LENGTH-1) {
      uint temp = (uint)((count_t)number / (count_t)steps[srciter]);
      curChar = md5pool[temp];
      number -= (count_t)((count_t)temp * (count_t)steps[srciter]);
		} else 
      curChar = md5pool[0];
    
    /* Encode current char for the destination word */
    nextArrayUint |= (curChar << shiftoffset);
    shiftoffset += 8;
    
    /* if we have packed 4 chars in the uint we have to write it to word */
    if (shiftoffset > 24) {
      getw(word, dstiter++) = nextArrayUint;
      shiftoffset = 0;
      nextArrayUint = 0;
    }
	}
  
  /* Append a single 1 bit after the message as needed by md5 */
  /* When arriving here shiftoffset is <= 24, so we can safely append one more char and encode it */
  nextArrayUint |= (0x80 << shiftoffset);
  getw(word, dstiter++) = nextArrayUint;
	
  /* zero the words padding */
  for (; dstiter < 16; ++dstiter)
  	getw(word, dstiter) = (uint)0;
	
	__syncthreads();
  
  /* write the message length in bits */
	getw(word, 14) = ORIGINAL_WORD_LENGTH * 8;
}

//////////////////////////////////////////////////////////////////////////////
/////////////       Ron Rivest's MD5 C Implementation       //////////////////
//////////////////////////////////////////////////////////////////////////////

/*
 **********************************************************************
 ** Copyright (C) 1990, RSA Data Security, Inc. All rights reserved. **
 **                                                                  **
 ** License to copy and use this software is granted provided that   **
 ** it is identified as the "RSA Data Security, Inc. MD5 Message     **
 ** Digest Algorithm" in all material mentioning or referencing this **
 ** software or this function.                                       **
 **                                                                  **
 ** License is also granted to make and use derivative works         **
 ** provided that such works are identified as "derived from the RSA **
 ** Data Security, Inc. MD5 Message Digest Algorithm" in all         **
 ** material mentioning or referencing the derived work.             **
 **                                                                  **
 ** RSA Data Security, Inc. makes no representations concerning      **
 ** either the merchantability of this software or the suitability   **
 ** of this software for any particular purpose.  It is provided "as **
 ** is" without express or implied warranty of any kind.             **
 **                                                                  **
 ** These notices must be retained in any copies of any part of this **
 ** documentation and/or software.                                   **
 **********************************************************************
 */


/* F, G and H are basic MD5 functions: selection, majority, parity */
#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z))) 

/* ROTATE_LEFT rotates x left n bits */
#define ROTATE_LEFT(x, n) (((x) << (n)) | ((x) >> (32-(n))))

/* FF, GG, HH, and II transformations for rounds 1, 2, 3, and 4 */
/* Rotation is separate from addition to prevent recomputation */
#define FF(a, b, c, d, x, s, ac) \
  {(a) += F ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define GG(a, b, c, d, x, s, ac) \
  {(a) += G ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define HH(a, b, c, d, x, s, ac) \
  {(a) += H ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define II(a, b, c, d, x, s, ac) \
  {(a) += I ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }


/* Basic MD5 step. Transform buf based on in.
 */
void inline __device__ md5_v2(const uint *in, uint &a, uint &b, uint &c, uint &d)
{
	const uint a0 = 0x67452301;
	const uint b0 = 0xEFCDAB89;
	const uint c0 = 0x98BADCFE;
	const uint d0 = 0x10325476;

	//Initialize hash value for this chunk:
	a = a0;
	b = b0;
	c = c0;
	d = d0;

  /* Round 1 */
#define S11 7
#define S12 12
#define S13 17
#define S14 22
  FF ( a, b, c, d, getw(in,  0), S11, 3614090360); /* 1 */
  FF ( d, a, b, c, getw(in,  1), S12, 3905402710); /* 2 */
  FF ( c, d, a, b, getw(in,  2), S13,  606105819); /* 3 */
  FF ( b, c, d, a, getw(in,  3), S14, 3250441966); /* 4 */
  FF ( a, b, c, d, getw(in,  4), S11, 4118548399); /* 5 */
  FF ( d, a, b, c, getw(in,  5), S12, 1200080426); /* 6 */
  FF ( c, d, a, b, getw(in,  6), S13, 2821735955); /* 7 */
  FF ( b, c, d, a, getw(in,  7), S14, 4249261313); /* 8 */
  FF ( a, b, c, d, getw(in,  8), S11, 1770035416); /* 9 */
  FF ( d, a, b, c, getw(in,  9), S12, 2336552879); /* 10 */
  FF ( c, d, a, b, getw(in, 10), S13, 4294925233); /* 11 */
  FF ( b, c, d, a, getw(in, 11), S14, 2304563134); /* 12 */
  FF ( a, b, c, d, getw(in, 12), S11, 1804603682); /* 13 */
  FF ( d, a, b, c, getw(in, 13), S12, 4254626195); /* 14 */
  FF ( c, d, a, b, getw(in, 14), S13, 2792965006); /* 15 */
  FF ( b, c, d, a, getw(in, 15), S14, 1236535329); /* 16 */
 
  /* Round 2 */
#define S21 5
#define S22 9
#define S23 14
#define S24 20
  GG ( a, b, c, d, getw(in,  1), S21, 4129170786); /* 17 */
  GG ( d, a, b, c, getw(in,  6), S22, 3225465664); /* 18 */
  GG ( c, d, a, b, getw(in, 11), S23,  643717713); /* 19 */
  GG ( b, c, d, a, getw(in,  0), S24, 3921069994); /* 20 */
  GG ( a, b, c, d, getw(in,  5), S21, 3593408605); /* 21 */
  GG ( d, a, b, c, getw(in, 10), S22,   38016083); /* 22 */
  GG ( c, d, a, b, getw(in, 15), S23, 3634488961); /* 23 */
  GG ( b, c, d, a, getw(in,  4), S24, 3889429448); /* 24 */
  GG ( a, b, c, d, getw(in,  9), S21,  568446438); /* 25 */
  GG ( d, a, b, c, getw(in, 14), S22, 3275163606); /* 26 */
  GG ( c, d, a, b, getw(in,  3), S23, 4107603335); /* 27 */
  GG ( b, c, d, a, getw(in,  8), S24, 1163531501); /* 28 */
  GG ( a, b, c, d, getw(in, 13), S21, 2850285829); /* 29 */
  GG ( d, a, b, c, getw(in,  2), S22, 4243563512); /* 30 */
  GG ( c, d, a, b, getw(in,  7), S23, 1735328473); /* 31 */
  GG ( b, c, d, a, getw(in, 12), S24, 2368359562); /* 32 */

  /* Round 3 */
#define S31 4
#define S32 11
#define S33 16
#define S34 23
  HH ( a, b, c, d, getw(in,  5), S31, 4294588738); /* 33 */
  HH ( d, a, b, c, getw(in,  8), S32, 2272392833); /* 34 */
  HH ( c, d, a, b, getw(in, 11), S33, 1839030562); /* 35 */
  HH ( b, c, d, a, getw(in, 14), S34, 4259657740); /* 36 */
  HH ( a, b, c, d, getw(in,  1), S31, 2763975236); /* 37 */
  HH ( d, a, b, c, getw(in,  4), S32, 1272893353); /* 38 */
  HH ( c, d, a, b, getw(in,  7), S33, 4139469664); /* 39 */
  HH ( b, c, d, a, getw(in, 10), S34, 3200236656); /* 40 */
  HH ( a, b, c, d, getw(in, 13), S31,  681279174); /* 41 */
  HH ( d, a, b, c, getw(in,  0), S32, 3936430074); /* 42 */
  HH ( c, d, a, b, getw(in,  3), S33, 3572445317); /* 43 */
  HH ( b, c, d, a, getw(in,  6), S34,   76029189); /* 44 */
  HH ( a, b, c, d, getw(in,  9), S31, 3654602809); /* 45 */
  HH ( d, a, b, c, getw(in, 12), S32, 3873151461); /* 46 */
  HH ( c, d, a, b, getw(in, 15), S33,  530742520); /* 47 */
  HH ( b, c, d, a, getw(in,  2), S34, 3299628645); /* 48 */

  /* Round 4 */
#define S41 6
#define S42 10
#define S43 15
#define S44 21
  II ( a, b, c, d, getw(in,  0), S41, 4096336452); /* 49 */
  II ( d, a, b, c, getw(in,  7), S42, 1126891415); /* 50 */
  II ( c, d, a, b, getw(in, 14), S43, 2878612391); /* 51 */
  II ( b, c, d, a, getw(in,  5), S44, 4237533241); /* 52 */
  II ( a, b, c, d, getw(in, 12), S41, 1700485571); /* 53 */
  II ( d, a, b, c, getw(in,  3), S42, 2399980690); /* 54 */
  II ( c, d, a, b, getw(in, 10), S43, 4293915773); /* 55 */
  II ( b, c, d, a, getw(in,  1), S44, 2240044497); /* 56 */
  II ( a, b, c, d, getw(in,  8), S41, 1873313359); /* 57 */
  II ( d, a, b, c, getw(in, 15), S42, 4264355552); /* 58 */
  II ( c, d, a, b, getw(in,  6), S43, 2734768916); /* 59 */
  II ( b, c, d, a, getw(in, 13), S44, 1309151649); /* 60 */
  II ( a, b, c, d, getw(in,  4), S41, 4149444226); /* 61 */
  II ( d, a, b, c, getw(in, 11), S42, 3174756917); /* 62 */
  II ( c, d, a, b, getw(in,  2), S43,  718787259); /* 63 */
  II ( b, c, d, a, getw(in,  9), S44, 3951481745); /* 64 */

	a += a0;
	b += b0;
	c += c0;
	d += d0;

}

//////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////

// The kernel (this is the entrypoint of GPU code)
// Calculates the 64-byte word from MD5POOL to be hashed in shared memory,
// calls the calculation routine, compares to target and flags if a match is found
extern "C"
__global__ void md5_search(ullong starting_number, uint words_per_call, uint iterations, ullong max_number, uint *succ, uint *target)
{
	count_t linidx = (count_t)(gridDim.x*blockIdx.y + blockIdx.x)*blockDim.x + threadIdx.x; // assuming blockDim.y = 1 and threadIdx.y = 0, always
	if(linidx >= words_per_call) { return; }
	linidx += (count_t)starting_number;

	/* get the shared memory region for our calculations */
	uint *word = &memory[0] + threadIdx.x*16;
	
	for (int i=0 ; i < iterations && linidx < max_number; ++i) {
		// calculate the dictionary word for this thread
		number2paddedword(linidx, word);

		// compute MD5 hash
		uint a, b, c, d;

		RSA_KERNEL(word, a, b, c, d);

		if(a == target[0] && b == target[1] && c == target[2] && d == target[3])
		{
			count_t *temp = (count_t *) &succ[0];
			*temp = linidx;
			succ[3] = 1;
			break;
		}
		__syncthreads();
		if (succ[3] != 0)
			break;
		
		linidx += (count_t)words_per_call;
	}
  /*
  succ[0] = target[0];
  succ[1] = target[1];
  succ[2] = target[2];
  succ[3] = target[3];
  */
}
