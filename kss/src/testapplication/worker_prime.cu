#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cutil.h"
#include <cutil_inline.h>
#include "worker_prime.h"

#define count_t ullong 

int __device__ isPrime(const ullong currentDivisor)
{
	ullong root = ((ullong)sqrt((double) currentDivisor))+1;
	
	for (ullong i=2; i< root; i++){
		/* in this case we are useless */
		if ((currentDivisor % i) == 0) {
			return 0;
		}
	}
	return 1;
}


void __device__ testCandidate(const ullong currentDivisor, ullong *currentSquareRoot, ullong *remainder, ullong *primes, ullong *exponents, uint *nextIndex)
{
	ullong oldRemainder = *remainder;
	ullong newRemainder = oldRemainder;
	ullong temp = oldRemainder;
	uint exponent = 0;
	
	while ((oldRemainder % currentDivisor) == 0) {
		newRemainder = oldRemainder / currentDivisor;
		
		/* write the calculated remainder - if this fails repeat the previous iteration with the new remainder */
		temp = atomicCAS(remainder, oldRemainder, newRemainder);
		if (temp != oldRemainder) {
			/* repeat previous iteration with the externally calculated new remainder */
			oldRemainder = temp;
		} else {
			/* our remainder got through, continue */
			exponent += 1;
			oldRemainder = newRemainder;
		}
	}
	
	if (exponent > 0) {
		/* get a unique index */
		uint ourIndex = 0;
		uint oldIndex;
get_an_index:
		oldIndex = *nextIndex;
		ourIndex = atomicCAS(nextIndex, oldIndex, oldIndex+1);
		
		/* if somebody intervened - try again */
		if (ourIndex != oldIndex)
			goto get_an_index;
		
		/* we have an index: write out result */
		primes[ourIndex] = currentDivisor;
		exponents[ourIndex] = exponent;
		
		/* write a root which is consistent with the new remainder */
write_new_root:
		oldRemainder = *remainder;
		temp = ceil(sqrt((double)(oldRemainder)));
		atomicExch(currentSquareRoot, temp);
		if (oldRemainder != *remainder)
			goto write_new_root;
	}
}

//////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////

// The kernel (this is the entrypoint of GPU code)
// calls the calculation routine, compares to target and flags if a match is found
extern "C"
__global__ void prime_factor(ullong currentDivisor, uint number_of_threads, uint iterations, uint *succ, ullong *primes, ullong *exponents, uint *nextIndex, ullong *currentSquareRoot, ullong *remainder)
{
	count_t linidx = (count_t)(gridDim.x*blockIdx.y + blockIdx.x)*blockDim.x + threadIdx.x; // assuming blockDim.y = 1 and threadIdx.y = 0, always
	if(linidx >= number_of_threads) { return; }
	linidx += (count_t)currentDivisor;
	
	uint have_to_avoid_multiples = 0;
	
	/* currentDivisor * 2 > currentDivisor + (number_of_threads * iterations)
	 * means that even the smallest possible multiple is outside this kernels scope.
	 * We can then skip the prime check because no thread processes the multiple of another.
	 */
	if (currentDivisor <= ((ullong) number_of_threads * (ullong)iterations))
		have_to_avoid_multiples = 1;

	for (int i=0 ; i < iterations && linidx < *currentSquareRoot && *remainder > 1 && *nextIndex < FACTORS_TO_FIND; ++i) {
		if (!have_to_avoid_multiples || isPrime(linidx))
			testCandidate(linidx, currentSquareRoot, remainder, primes, exponents, nextIndex);

		__syncthreads();
		linidx += (count_t)number_of_threads;
	}
	
	/* correct last loop */
	linidx -= (count_t)number_of_threads;
	
	__syncthreads();
	if (*nextIndex >= FACTORS_TO_FIND || *remainder <= 1 || linidx > *currentSquareRoot)
		*succ = 1;
}
