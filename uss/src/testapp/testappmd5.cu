#include "hip/hip_runtime.h"
/*
 * this is a testapplication filling using the
 * library calls offered by uss_library
 */
 
 /*
  * CURRENT EXAMPLE
  * MD5 cracking
  *
  * start with: testappmd5 <benchmark-id> <target-value>
  *
  */
  
//USS
#include "../library/uss.h"
#include <hip/hip_runtime.h>

#define TESTDEBUG 1
#define BENCHMARK_MAIN 1

#if(BENCHMARK_MAIN == 1)
#include "../benchmark/dwatch.h"
#endif

//////////////////////////////////////////////
//											//
// own user-defined USS structures			//
//											//
//////////////////////////////////////////////
//               0         1         2         3         4         5         6 
//               01234567890123456789012345678901234567890123456789012345678901
//#define MD5POOL "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789"
//#define MD5POOL "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ"
#define MD5POOL "abcdefghijklmnopqrstuvwxyz0123456789"
#define ORIGINAL_WORD_LENGTH 6
#define WORDS_PER_BATCH 500
//#define WORDS_PER_BATCH_GPU 1000000000
//#define WORDS_PER_BATCH_GPU 100000000
#define WORDS_PER_BATCH_GPU 1000000

#define NUM_OF_CHARS (sizeof(MD5POOL) - 1)

using namespace std;

typedef unsigned int uint;
typedef unsigned long long ullong;

#include <stdio.h>
#include <stdlib.h>

#include <string>
#include <string.h>
#include <iostream>
#include <math.h>
#include <cutil.h>
#include <cutil_inline.h>

#define for_each_word_position_i for (int i=0; i<ORIGINAL_WORD_LENGTH; ++i)

#define uint4_type unsigned int
typedef unsigned int size_type; // must be 32bit
typedef unsigned char md5uint1; //  8bit
typedef uint4_type    md5uint4;  // 32bit
  
//////////////////////////////////////////////
//											//
// header like additional definitions		//
//											//
//////////////////////////////////////////////


//define your personal meta_data and meta_checkpoint
struct meta_checkpoint
{
	int is_finished;
	signed long *workstatus;
	
	std::string hash_to_search;
	uint4_type raw_hash_to_search[4];
	
	unsigned long long currentWordNumber;
	bool foundsolution;
	
	int shared_mem_available;
	hipDeviceptr_t dev_success;
	hipDeviceptr_t dev_target;
	hipStream_t cudastream;
	hipCtx_t cudaContext;
	hipFunction_t cudaFunction;
	hipModule_t cudaModule;
};
  
struct meta_data
{
	int empty;
};

enum {blocksize = 64}; // VC6 won't eat a const static int here
bool finalized;
md5uint1 buffer[blocksize]; // bytes that didn't fit in last 64 byte chunk
md5uint4 count[2];   // 64bit counter for number of bits (lo, hi)
md5uint4 state[4];   // digest so far
md5uint1 digest[16]; // the result

void init();
void transform(const md5uint1 block[blocksize]);
static void decode(md5uint4 output[], const md5uint1 input[], size_type len);
static void encode(md5uint1 output[], const md5uint4 input[], size_type len);
void update(const unsigned char *buf, size_type length);
void update(const char *buf, size_type length);
void finalize();
string hexdigest();	 
//////////////////////////////////////////////
//											//
// helpers for all implementations			//
//											//
//////////////////////////////////////////////
// Constants for MD5Transform routine.
#define S11 7
#define S12 12
#define S13 17
#define S14 22
#define S21 5
#define S22 9
#define S23 14
#define S24 20
#define S31 4
#define S32 11
#define S33 16
#define S34 23
#define S41 6
#define S42 10
#define S43 15
#define S44 21

///////////////////////////////////////////////

// F, G, H and I are basic MD5 functions.
inline md5uint4 F(md5uint4 x, md5uint4 y, md5uint4 z) {
  return x&y | ~x&z;
}

inline md5uint4 G(md5uint4 x, md5uint4 y, md5uint4 z) {
  return x&z | y&~z;
}

inline md5uint4 H(md5uint4 x, md5uint4 y, md5uint4 z) {
  return x^y^z;
}

inline md5uint4 I(md5uint4 x, md5uint4 y, md5uint4 z) {
  return y ^ (x | ~z);
}

// rotate_left rotates x left n bits.
inline md5uint4 rotate_left(md5uint4 x, int n) {
  return (x << n) | (x >> (32-n));
}

// FF, GG, HH, and II transformations for rounds 1, 2, 3, and 4.
// Rotation is separate from addition to prevent recomputation.
inline void FF(md5uint4 &a, md5uint4 b, md5uint4 c, md5uint4 d, md5uint4 x, md5uint4 s, md5uint4 ac) {
  a = rotate_left(a+ F(b,c,d) + x + ac, s) + b;
}

inline void GG(md5uint4 &a, md5uint4 b, md5uint4 c, md5uint4 d, md5uint4 x, md5uint4 s, md5uint4 ac) {
  a = rotate_left(a + G(b,c,d) + x + ac, s) + b;
}

inline void HH(md5uint4 &a, md5uint4 b, md5uint4 c, md5uint4 d, md5uint4 x, md5uint4 s, md5uint4 ac) {
  a = rotate_left(a + H(b,c,d) + x + ac, s) + b;
}

inline void II(md5uint4 &a, md5uint4 b, md5uint4 c, md5uint4 d, md5uint4 x, md5uint4 s, md5uint4 ac) {
  a = rotate_left(a + I(b,c,d) + x + ac, s) + b;
}

 
//////////////////////////////////////////////
//											//
// CPU implementation						//
//											//
//////////////////////////////////////////////
bool nextIndexWord(char* word, int len)
{
  int maxsize = NUM_OF_CHARS - 1;
  bool overflow = false;
  
  /* increase from lowest to highest bit */
  for (int i = len - 1; i >= 0; --i) {
    overflow = false;
    if (word[i] < maxsize) {
      word[i]++;
    } else {
      word[i] = 0;
      overflow = true;
    }
    /* stop if all overflow has been accounted for */
    if (!overflow)
      break;
  }
  /* if even the first bit overflowed we have reached the end of our range... */
  return !overflow;
}

/* 
 * NOTE: This function will fail badly if it is called with a number >= steps[0] * NUM_OF_CHARS
 */
void number2indexword (unsigned long long number, char *word, int id)
{
	uint x=0;
	unsigned long long temp, step;
	step = (unsigned long long) pow(NUM_OF_CHARS, (ORIGINAL_WORD_LENGTH));
	if (number > step) {
		printf("Fatal error! Number %lld is too large", number);
		exit(-1);
	}
	step /= NUM_OF_CHARS;

	/*
	 * Special case: Length of words is 0 or 1
	 * These cases can be determined at compile time and can therefore
	 * be optimized away by the compiler
	 */
	if (ORIGINAL_WORD_LENGTH < 1)
		return;
	else if (ORIGINAL_WORD_LENGTH == 1) {
		word[x] = number;
		return;
	}
	
	word[x] = 0;
	while (1) {
		if (number <= step) {
			x++;
			step /= NUM_OF_CHARS;
			if (x >= ORIGINAL_WORD_LENGTH-1)
				break;
			word[x] = 0;
			continue;
		}
		temp = (uint)floor(number/step);
		word[x] = temp;
		number -= ((unsigned long long) temp * step);
	}
	word[ORIGINAL_WORD_LENGTH-1] = number;
}

int myalgo_cpu_init(void *md_void, void *mcp_void, int device_id)
{
	//no init for CPU
#if(TESTDEBUG == 1)
	printf("myalgo_CPU_init()\n");
#endif
	return 0;
}

int myalgo_cpu_main(void *md_void, void *mcp_void, int device_id)
{
#if(TESTDEBUG == 1)
	printf("myalgo_CPU_main()");
#endif
	//struct meta_data *md = (struct meta_data*) md_void;
	struct meta_checkpoint *resources = (struct meta_checkpoint*) mcp_void;

	char md5pool[sizeof(MD5POOL)] = MD5POOL;
	char currentBruteforceIndex[ORIGINAL_WORD_LENGTH+1];
	char currentword[ORIGINAL_WORD_LENGTH+1];
	currentword[ORIGINAL_WORD_LENGTH] = 0;
	currentBruteforceIndex[ORIGINAL_WORD_LENGTH] = 0;
	std::string currentmd5;
	bool bWordsleft = false;

	*(resources->workstatus) = true;
	/* initialize the current index word to the correct number */
	number2indexword(resources->currentWordNumber, &currentBruteforceIndex[0], /*Daniel: replaced unused this->id*/ 0);

	for (int words = 0; words < WORDS_PER_BATCH && *(resources->workstatus); ++words)
	{
	init();
	for (int i=0; i<ORIGINAL_WORD_LENGTH; ++i)
	{
	  currentword[i] = md5pool[currentBruteforceIndex[i]];
	}
	update(currentword, sizeof(currentword)-1);
	finalize();
	currentmd5 = hexdigest();
	bWordsleft = nextIndexWord(&(currentBruteforceIndex[0]), ORIGINAL_WORD_LENGTH);
	resources->currentWordNumber++;
	*(resources->workstatus) = (bWordsleft && resources->hash_to_search != currentmd5);
	}
	if (resources->currentWordNumber % 5000000 == 0)
	{
		#if(TESTDEBUG == 1)
		printf("Currently working on cpu with ( %lld ) \n", resources->currentWordNumber);
		#endif
	}
	if (!*(resources->workstatus) && bWordsleft)
	{
		#if(TESTDEBUG == 1)
		printf("Found solution -string- ( %lld )\n", resources->currentWordNumber);
		#endif
		resources->foundsolution = true;
		resources->is_finished = 1;
	} else if (bWordsleft)
		*(resources->workstatus) = resources->currentWordNumber;
		
	return 0;
}

 int myalgo_cpu_free(void *md_void, void *mcp_void, int device_id)
 {
	//no cleanup for CPU
#if(TESTDEBUG == 1)	
	printf("myalgo_CPU_free()\n");
#endif
	return 0;
 }


//////////////////////////////////////////////
//											//
// CUDA	implementation						//
//											//
//////////////////////////////////////////////
//
// MD5 magic numbers. These will be loaded into on-device "constant" memory
//
static const uint k_cpu[64] =
{
  0xd76aa478, 	0xe8c7b756,	0x242070db,	0xc1bdceee,
  0xf57c0faf,	0x4787c62a, 	0xa8304613,	0xfd469501,
  0x698098d8,	0x8b44f7af,	0xffff5bb1,	0x895cd7be,
  0x6b901122, 	0xfd987193, 	0xa679438e,	0x49b40821,

  0xf61e2562,	0xc040b340, 	0x265e5a51, 	0xe9b6c7aa,
  0xd62f105d,	0x2441453,	0xd8a1e681,	0xe7d3fbc8,
  0x21e1cde6,	0xc33707d6, 	0xf4d50d87, 	0x455a14ed,
  0xa9e3e905,	0xfcefa3f8, 	0x676f02d9, 	0x8d2a4c8a,

  0xfffa3942,	0x8771f681, 	0x6d9d6122, 	0xfde5380c,
  0xa4beea44, 	0x4bdecfa9, 	0xf6bb4b60, 	0xbebfbc70,
  0x289b7ec6, 	0xeaa127fa, 	0xd4ef3085,	0x4881d05,
  0xd9d4d039, 	0xe6db99e5, 	0x1fa27cf8, 	0xc4ac5665,

  0xf4292244, 	0x432aff97, 	0xab9423a7, 	0xfc93a039,
  0x655b59c3, 	0x8f0ccc92, 	0xffeff47d, 	0x85845dd1,
  0x6fa87e4f, 	0xfe2ce6e0, 	0xa3014314, 	0x4e0811a1,
  0xf7537e82, 	0xbd3af235, 	0x2ad7d2bb, 	0xeb86d391,
};

static const uint rconst_cpu[16] =
{
  7, 12, 17, 22,   5,  9, 14, 20,   4, 11, 16, 23,   6, 10, 15, 21
};


/* Function to copy over the constants to the gpu */
void init_constants(hipModule_t *cudaModule)
{
  hipDeviceptr_t dptr;
  /*Daniel changed unsigned int bytes to size_t bytes (maybe CUDA4.0 wants this?)*/
  size_t bytes;

  cutilDrvSafeCall( hipModuleGetGlobal( &dptr, &bytes, *cudaModule, "k" ));
  cutilDrvSafeCall( hipMemcpyHtoD(dptr,  k_cpu,  bytes));

  cutilDrvSafeCall( hipModuleGetGlobal( &dptr, &bytes, *cudaModule, "rconst" ));
  cutilDrvSafeCall( hipMemcpyHtoD(dptr,  rconst_cpu,  bytes));

  if (ORIGINAL_WORD_LENGTH > 0) {
    uint steps_cpu[ORIGINAL_WORD_LENGTH];
    steps_cpu[ORIGINAL_WORD_LENGTH-1] = 1;
    for (int i = ORIGINAL_WORD_LENGTH-2; i >= 0; --i)
      steps_cpu[i] = steps_cpu[i+1] * NUM_OF_CHARS;

    cutilDrvSafeCall( hipModuleGetGlobal( &dptr, &bytes, *cudaModule, "steps" ));
    cutilDrvSafeCall( hipMemcpyHtoD(dptr,  steps_cpu,  bytes));
  }
}

/* A helper to export the kernel call to C++ code not compiled with nvcc */
static void execute_kernel(unsigned long long starting_number, struct meta_checkpoint *resources)
{
  //
  // The icky part: compute the optimal number of threads per block,
  // and the number of blocks
  //
  int dynShmemPerThread = 64;	// built in the algorithm
  int staticShmemPerBlock = 32;	// read from .cubin file
  //int words_per_call = 25000; // just a guess of what is efficiently possible
  int words_per_call = 1000000; // just a guess of what is efficiently possible
	int iterations = ceil(WORDS_PER_BATCH_GPU / words_per_call);

  int threads_per_block = min((double)512, floor((double)(resources->shared_mem_available - staticShmemPerBlock) / (double)dynShmemPerThread));
  int blocks_in_grid = ceil((double)words_per_call / (double)threads_per_block);

  //printf("words_per_call = %d, iterations = %d, threads_per_block = %d, blocks_in_grid = %d\n", words_per_call, iterations, threads_per_block, blocks_in_grid);

  // set kernel parameters
  // md5_search(uint starting_number, uint words_per_call, uint max_number, uint *succ)
  int offset = 0;
  offset = (offset + __alignof(starting_number) - 1) & ~(__alignof(starting_number) - 1); // adjust offset to meet alignment requirement
  cutilDrvSafeCallNoSync(cuParamSetv(resources->cudaFunction, offset, &starting_number, sizeof(starting_number)));
  offset += sizeof(starting_number);

  offset = (offset + __alignof(words_per_call) - 1) & ~(__alignof(words_per_call) - 1);
  cutilDrvSafeCall(cuParamSeti(resources->cudaFunction, offset, words_per_call));
  offset += sizeof(words_per_call);

  offset = (offset + __alignof(iterations) - 1) & ~(__alignof(iterations) - 1);
  cutilDrvSafeCall(cuParamSeti(resources->cudaFunction, offset, iterations));
  offset += sizeof(iterations);

  unsigned long long max_number = (unsigned long long) pow(NUM_OF_CHARS, ORIGINAL_WORD_LENGTH);
  offset = (offset + __alignof(max_number) - 1) & ~(__alignof(max_number) - 1);
  cutilDrvSafeCallNoSync(cuParamSetv(resources->cudaFunction, offset, &max_number, sizeof(max_number)));
  offset += sizeof(max_number);

  void* ptr = (void*)(size_t)resources->dev_success;
  offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
  cutilDrvSafeCallNoSync(cuParamSetv(resources->cudaFunction, offset, &ptr, sizeof(ptr)));
  offset += sizeof(ptr);

  ptr = (void*)(size_t)resources->dev_target;
  offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
  cutilDrvSafeCallNoSync(cuParamSetv(resources->cudaFunction, offset, &ptr, sizeof(ptr)));
  offset += sizeof(ptr);

  cutilDrvSafeCall(cuParamSetSize(resources->cudaFunction, offset));

  // set execution configuration
  dim3 block;
  block.x = threads_per_block; block.y = 1;
  cutilDrvSafeCall(cuFuncSetBlockShape(resources->cudaFunction, block.x, block.y, 1 ));

  dim3 grid;
  grid.x = blocks_in_grid; grid.y = 1;

  cutilDrvSafeCall(cuFuncSetSharedSize (resources->cudaFunction, threads_per_block*dynShmemPerThread));
  
  //cout << "Executing with " << starting_number << ", " << words_per_call << ", " << max_number << ", " << resources->dev_success << ", " << resources->dev_target << "\n";
  /* Launch the kernel */
  cutilDrvSafeCall(cuLaunchGridAsync( resources->cudaFunction, grid.x, grid.y, resources->cudastream ));

  /* Wait for its completion */
  //cutilDrvSafeCall( hipStreamSynchronize(resources->cudastream) );
	cutilDrvSafeCall( hipCtxSynchronize() );
}



int myalgo_cuda_init(void *md_void, void *mcp_void, int device_id)
{
#if(TESTDEBUG == 1)
	printf("myalgo_CUDA_init()\n");
#endif
	//struct meta_data *md = (struct meta_data*) md_void;
	struct meta_checkpoint *resources = (struct meta_checkpoint*) mcp_void;

  // Initialize the driver API
  hipDevice_t cudaDevice;
  hipError_t status;
  hipInit(0);
  const char *step;
  
  /* Get the available number of shared memory per block */
  step = "hipDeviceGetAttribute";
  /*Daniel: replaces cu->api_device_number with device_id (this is my notation of which GPU to run on)*/
  if (hipDeviceGetAttribute( &resources->shared_mem_available, hipDeviceAttributeMaxSharedMemoryPerBlock, device_id ) != hipSuccess)
    goto cleanup_and_fail;
  
  /* Create a context on the correct device */
  step = "hipDeviceGet";
  cutilDrvSafeCallNoSync(hipDeviceGet(&cudaDevice, device_id));
  step = "hipCtxCreate";
  if ((status = hipCtxCreate( &resources->cudaContext, /* ommitting bof error: hipDeviceScheduleBlockingSync | hipDeviceScheduleYield*/ 0, cudaDevice )) != hipSuccess)
    goto cleanup_and_fail;

  /* Get the pointer to the function inside the cubin */
  step = "hipModuleLoad";
  if ((status = hipModuleLoad(&resources->cudaModule, "./testapp/md5.cubin")) != hipSuccess)
    goto cleanup_and_fail;
  step = "hipModuleGetFunction";
  if ((status = hipModuleGetFunction(&resources->cudaFunction, resources->cudaModule, "md5_search")) != hipSuccess)
    goto cleanup_and_fail;

  /* create a stream for async operations */
  cutilDrvSafeCall( hipStreamCreateWithFlags (&resources->cudastream, 0) );
  
  /* copy over the constants */
  init_constants(&resources->cudaModule);
  
  // allocate GPU memory for match signal
  step = "MemAlloc & set";
  cutilDrvSafeCall( hipMalloc( &resources->dev_success, 4*sizeof(uint)));
  cutilDrvSafeCall( hipMemsetD8( resources->dev_success, 0, 4*sizeof(uint)));
  cutilDrvSafeCall( hipMalloc( &resources->dev_target, 4*sizeof(uint)));
  cutilDrvSafeCall( hipMemcpyHtoD (resources->dev_target, &resources->raw_hash_to_search[0], 4*sizeof(uint)));
  
	return 0;

cleanup_and_fail:
	printf("cudainitFunc() failed at %s with error %i\n", step, (int)status);
  cutilDrvSafeCall(cuCtxDetach(resources->cudaContext));

	return -1;
}
 
int myalgo_cuda_main(void *md_void, void *mcp_void, int device_id)
{
	#if(TESTDEBUG == 1)
	printf("myalgo_CUDA_main()\n");
	#endif
	struct meta_checkpoint *resources = (struct meta_checkpoint*) mcp_void;

	*(resources->workstatus) = true;
	bool bWordsleft = false;

	execute_kernel(resources->currentWordNumber, resources);
	resources->currentWordNumber += WORDS_PER_BATCH_GPU;
	#if(TESTDEBUG == 1)
	printf("Currently working on cuda ( %lld )\n", resources->currentWordNumber);
	#endif
	bWordsleft = (resources->currentWordNumber < pow(NUM_OF_CHARS, ORIGINAL_WORD_LENGTH));
	*(resources->workstatus) = (bWordsleft);
	if (bWordsleft)
		*(resources->workstatus) = resources->currentWordNumber;

	uint ret[4];
	cutilDrvSafeCall( hipMemcpyDtoH (ret, resources->dev_success, sizeof(uint)*4));

	if (ret[3]) {
	/* word has been found */
	*(resources->workstatus) = false;
	char md5pool[sizeof(MD5POOL)] = MD5POOL;
	char currentword[ORIGINAL_WORD_LENGTH+1];
	currentword[ORIGINAL_WORD_LENGTH] = 0;
		unsigned long long *temp = (unsigned long long *) &ret[0];
	number2indexword(*temp, &currentword[0], /*Daniel: replaced unused this->id*/ 0);
	//    number2indexword(resources->currentWordNumber-1, &currentword[0]);
	for_each_word_position_i
	  currentword[i] = md5pool[currentword[i]];

	#if(TESTDEBUG == 1)
	printf("Found solution -string- ( &llu )", *temp);
	#endif
	resources->foundsolution = true;
	resources->is_finished = 1;
	}

	return 0;
}
 
int myalgo_cuda_free(void *md_void, void *mcp_void, int device_id)
{
#if(TESTDEBUG == 1)
	printf("myalgo_CUDA_free()\n");
#endif
	struct meta_checkpoint *resources = (struct meta_checkpoint*) mcp_void;

	cutilDrvSafeCall(hipFree(resources->dev_success));
	cutilDrvSafeCall(hipFree(resources->dev_target));

	cutilDrvSafeCall(hipModuleUnload(resources->cudaModule));
	cutilDrvSafeCall(hipStreamDestroy(resources->cudastream) );
	cutilDrvSafeCall(hipCtxDestroy(resources->cudaContext));

	return 0;
}

//////////////////////////////////////////////
//											//
// methods for preparating data for main() 	//
//											//
//////////////////////////////////////////////
void init()
{
  finalized=false;

  count[0] = 0;
  count[1] = 0;

  // load magic initialization constants.
  state[0] = 0x67452301;
  state[1] = 0xefcdab89;
  state[2] = 0x98badcfe;
  state[3] = 0x10325476;
}

//////////////////////////////

// decodes input (unsigned char) into output (Worker_md5::md5uint4). Assumes len is a multiple of 4.
void decode(md5uint4 output[], const md5uint1 input[], size_type len)
{
  for (unsigned int i = 0, j = 0; j < len; i++, j += 4)
    output[i] = ((md5uint4)input[j]) | (((md5uint4)input[j+1]) << 8) |
      (((md5uint4)input[j+2]) << 16) | (((md5uint4)input[j+3]) << 24);
}

//////////////////////////////

// encodes input (Worker_md5::md5uint4) into output (unsigned char). Assumes len is
// a multiple of 4.
void encode(md5uint1 output[], const md5uint4 input[], size_type len)
{
  for (size_type i = 0, j = 0; j < len; i++, j += 4) {
    output[j] = input[i] & 0xff;
    output[j+1] = (input[i] >> 8) & 0xff;
    output[j+2] = (input[i] >> 16) & 0xff;
    output[j+3] = (input[i] >> 24) & 0xff;
  }
}

// apply MD5 algo on a block
void transform(const md5uint1 block[blocksize])
{
  md5uint4 a = state[0], b = state[1], c = state[2], d = state[3], x[16];
  decode (x, block, blocksize);
  
  /* Round 1 */
  FF (a, b, c, d, x[ 0], S11, 0xd76aa478); /* 1 */
  FF (d, a, b, c, x[ 1], S12, 0xe8c7b756); /* 2 */
  FF (c, d, a, b, x[ 2], S13, 0x242070db); /* 3 */
  FF (b, c, d, a, x[ 3], S14, 0xc1bdceee); /* 4 */
  FF (a, b, c, d, x[ 4], S11, 0xf57c0faf); /* 5 */
  FF (d, a, b, c, x[ 5], S12, 0x4787c62a); /* 6 */
  FF (c, d, a, b, x[ 6], S13, 0xa8304613); /* 7 */
  FF (b, c, d, a, x[ 7], S14, 0xfd469501); /* 8 */
  FF (a, b, c, d, x[ 8], S11, 0x698098d8); /* 9 */
  FF (d, a, b, c, x[ 9], S12, 0x8b44f7af); /* 10 */
  FF (c, d, a, b, x[10], S13, 0xffff5bb1); /* 11 */
  FF (b, c, d, a, x[11], S14, 0x895cd7be); /* 12 */
  FF (a, b, c, d, x[12], S11, 0x6b901122); /* 13 */
  FF (d, a, b, c, x[13], S12, 0xfd987193); /* 14 */
  FF (c, d, a, b, x[14], S13, 0xa679438e); /* 15 */
  FF (b, c, d, a, x[15], S14, 0x49b40821); /* 16 */

  /* Round 2 */
  GG (a, b, c, d, x[ 1], S21, 0xf61e2562); /* 17 */
  GG (d, a, b, c, x[ 6], S22, 0xc040b340); /* 18 */
  GG (c, d, a, b, x[11], S23, 0x265e5a51); /* 19 */
  GG (b, c, d, a, x[ 0], S24, 0xe9b6c7aa); /* 20 */
  GG (a, b, c, d, x[ 5], S21, 0xd62f105d); /* 21 */
  GG (d, a, b, c, x[10], S22,  0x2441453); /* 22 */
  GG (c, d, a, b, x[15], S23, 0xd8a1e681); /* 23 */
  GG (b, c, d, a, x[ 4], S24, 0xe7d3fbc8); /* 24 */
  GG (a, b, c, d, x[ 9], S21, 0x21e1cde6); /* 25 */
  GG (d, a, b, c, x[14], S22, 0xc33707d6); /* 26 */
  GG (c, d, a, b, x[ 3], S23, 0xf4d50d87); /* 27 */
  GG (b, c, d, a, x[ 8], S24, 0x455a14ed); /* 28 */
  GG (a, b, c, d, x[13], S21, 0xa9e3e905); /* 29 */
  GG (d, a, b, c, x[ 2], S22, 0xfcefa3f8); /* 30 */
  GG (c, d, a, b, x[ 7], S23, 0x676f02d9); /* 31 */
  GG (b, c, d, a, x[12], S24, 0x8d2a4c8a); /* 32 */

  /* Round 3 */
  HH (a, b, c, d, x[ 5], S31, 0xfffa3942); /* 33 */
  HH (d, a, b, c, x[ 8], S32, 0x8771f681); /* 34 */
  HH (c, d, a, b, x[11], S33, 0x6d9d6122); /* 35 */
  HH (b, c, d, a, x[14], S34, 0xfde5380c); /* 36 */
  HH (a, b, c, d, x[ 1], S31, 0xa4beea44); /* 37 */
  HH (d, a, b, c, x[ 4], S32, 0x4bdecfa9); /* 38 */
  HH (c, d, a, b, x[ 7], S33, 0xf6bb4b60); /* 39 */
  HH (b, c, d, a, x[10], S34, 0xbebfbc70); /* 40 */
  HH (a, b, c, d, x[13], S31, 0x289b7ec6); /* 41 */
  HH (d, a, b, c, x[ 0], S32, 0xeaa127fa); /* 42 */
  HH (c, d, a, b, x[ 3], S33, 0xd4ef3085); /* 43 */
  HH (b, c, d, a, x[ 6], S34,  0x4881d05); /* 44 */
  HH (a, b, c, d, x[ 9], S31, 0xd9d4d039); /* 45 */
  HH (d, a, b, c, x[12], S32, 0xe6db99e5); /* 46 */
  HH (c, d, a, b, x[15], S33, 0x1fa27cf8); /* 47 */
  HH (b, c, d, a, x[ 2], S34, 0xc4ac5665); /* 48 */

  /* Round 4 */
  II (a, b, c, d, x[ 0], S41, 0xf4292244); /* 49 */
  II (d, a, b, c, x[ 7], S42, 0x432aff97); /* 50 */
  II (c, d, a, b, x[14], S43, 0xab9423a7); /* 51 */
  II (b, c, d, a, x[ 5], S44, 0xfc93a039); /* 52 */
  II (a, b, c, d, x[12], S41, 0x655b59c3); /* 53 */
  II (d, a, b, c, x[ 3], S42, 0x8f0ccc92); /* 54 */
  II (c, d, a, b, x[10], S43, 0xffeff47d); /* 55 */
  II (b, c, d, a, x[ 1], S44, 0x85845dd1); /* 56 */
  II (a, b, c, d, x[ 8], S41, 0x6fa87e4f); /* 57 */
  II (d, a, b, c, x[15], S42, 0xfe2ce6e0); /* 58 */
  II (c, d, a, b, x[ 6], S43, 0xa3014314); /* 59 */
  II (b, c, d, a, x[13], S44, 0x4e0811a1); /* 60 */
  II (a, b, c, d, x[ 4], S41, 0xf7537e82); /* 61 */
  II (d, a, b, c, x[11], S42, 0xbd3af235); /* 62 */
  II (c, d, a, b, x[ 2], S43, 0x2ad7d2bb); /* 63 */
  II (b, c, d, a, x[ 9], S44, 0xeb86d391); /* 64 */

  state[0] += a;
  state[1] += b;
  state[2] += c;
  state[3] += d;

  // Zeroize sensitive information.
  memset(x, 0, sizeof x);
}

// MD5 block update operation. Continues an MD5 message-digest
// operation, processing another message block
void update(const unsigned char input[], size_type length)
{
  // compute number of bytes mod 64
  size_type index = count[0] / 8 % blocksize;

  // Update number of bits
  if ((count[0] += (length << 3)) < (length << 3))
    count[1]++;
  count[1] += (length >> 29);

  // number of bytes we need to fill in buffer
  size_type firstpart = 64 - index;

  size_type i;

  // transform as many times as possible.
  if (length >= firstpart)
  {
    // fill buffer first, transform
    memcpy(&buffer[index], input, firstpart);
    transform(buffer);

    // transform chunks of blocksize (64 bytes)
    for (i = firstpart; i + blocksize <= length; i += blocksize)
      transform(&input[i]);

    index = 0;
  }
  else
    i = 0;

  // buffer remaining input
  memcpy(&buffer[index], &input[i], length-i);
}

// for convenience provide a verson with signed char
void update(const char input[], size_type length)
{
  update((const unsigned char*)input, length);
}

// MD5 finalization. Ends an MD5 message-digest operation, writing the
// the message digest and zeroizing the context.
void finalize()
{
  static unsigned char padding[64] = {
    0x80, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0
  };

  if (!finalized) {
    // Save number of bits
    unsigned char bits[8];
    encode(bits, count, 8);

    // pad out to 56 mod 64.
    size_type index = count[0] / 8 % 64;
    size_type padLen = (index < 56) ? (56 - index) : (120 - index);
    update(padding, padLen);

    // Append length (before padding)
    update(bits, 8);
    
    // Store state in digest
    encode(digest, state, 16);

    // Zeroize sensitive information.
    memset(buffer, 0, sizeof buffer);
    memset(count, 0, sizeof count);

    finalized=true;
  }
}

// return hex representation of digest as string
string hexdigest()
{
  if (!finalized)
    return "";

  char buf[33];
  for (int i=0; i<16; i++)
    sprintf(buf+i*2, "%02x", digest[i]);
  buf[32]=0;

  return std::string(buf);
}
 
//////////////////////////////////////////////
//											//
// MAIN (fills msi and calls libuss_start)	//
//											//
//////////////////////////////////////////////
int main(int argc, char *argv[])
{
/***************************************\
* parse testapp input					*
\***************************************/
	int id = 1;
	int modifier; //determines length
	if(argc == 1)
	{
		//default mode
		printf("<<< test application for uss_library>>>\n");
		//md5 just needs id and this is 1
	}
	else if(argc == 2)
	{
		//tobias mode (modifier is set below)
		id = atoi(argv[1]);
		//modifier = id;
		//md5 just needs id because it produces its keys depending on id
	}
	else if(argc == 3)
	{
		//tobias mode (modifier is set below)
		id = atoi(argv[1]);
		//modifier = atoi(argv[2]);
		//md5 just needs id because it produces its keys depending on id
	}
	else
	{
		printf("bad nof input parameters\n");
		exit(-1);
	}
	
	#if(BENCHMARK_MAIN == 1)
	init_dwatch();
	#endif
	
/***************************************\
* fill meta sched info (MSI) struct		*
\***************************************/
	int ret = 0;
	struct meta_sched_info msi;
	memset(&msi, 0, sizeof(struct meta_sched_info));
	
	//insert USS_ACCEL_TYPE_CPU
	ret = libuss_fill_msi(&msi, USS_ACCEL_TYPE_CPU, 2, 0, 
					&myalgo_cpu_init, 
					&myalgo_cpu_main, 
					&myalgo_cpu_free);
	if(ret == -1) {printf("Error with malloc\n"); return -1;}
	

	//insert USS_ACCEL_TYPE_CUDA
	ret = libuss_fill_msi(&msi, USS_ACCEL_TYPE_CUDA, 3, 0, 
					&myalgo_cuda_init, 
					&myalgo_cuda_main, 
					&myalgo_cuda_free);
	if(ret == -1) {printf("Error with malloc\n"); return -1;}

/***************************************\
* fill meta checkpoint (MCP)			*
\***************************************/
	struct meta_checkpoint resources;

	resources.foundsolution = false;
	resources.currentWordNumber = 0;

	init();
	int r;
	char md5pool[sizeof(MD5POOL)] = MD5POOL;
	char originalword[ORIGINAL_WORD_LENGTH+1];

	/* even id's get to the end of the md5pool, odd ones to the beginning */
	modifier = (id + ((id % 2) * (-2) * id)) * (-1); //modifier is -id or id
	//modifier = -1 * id;
	r = (NUM_OF_CHARS - 1 + modifier) % NUM_OF_CHARS;
	while (r < 0) r+= NUM_OF_CHARS;

	for_each_word_position_i
	{
		originalword[i] = md5pool[r];
		r = (r + 1) % NUM_OF_CHARS;
	}
	
	originalword[ORIGINAL_WORD_LENGTH] = 0;
	update(originalword, sizeof(originalword)-1);
	finalize();
	resources.hash_to_search = hexdigest();
	for (int i=0; i<4; ++i)
		resources.raw_hash_to_search[i] = state[i];

	/*this variable by tobias is design to hold the status
	 *
	 */
	signed long workstatus = true;
	resources.workstatus = &workstatus;
	
	/*this variable is my own finished indicator
	 *0=still work to do
	 *1=finished
	 */
	resources.is_finished = 0;
	
/***************************************\
* fill meta data (MD)					*
\***************************************/
	struct meta_data md;
	//is empty here
	
/***************************************\
* call libuss_start to enter			*
\***************************************/
	int run_on;
	int device_id;
	libuss_start(&msi, (void*)&md, (void*)&resources, &(resources.is_finished), &run_on, &device_id);
	
/***************************************\
* free meta sched info (MSI) memory		*
\***************************************/
	libuss_free_msi(&msi);

	#if(BENCHMARK_MAIN == 1)
	//returns id and total turnaround time in ms
	printf("%i %ld\n", id, (long)diff_dwatch());
	#endif
	return 0;
 }
