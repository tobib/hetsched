#include "hip/hip_runtime.h"
/*
 * this is a testapplication filling using the
 * library calls offered by uss_library
 */
 
 /*
  * CURRENT EXAMPLE
  * prime factorization
  *
  * start with: testappprime <benchmark-id> <target-value>
  *
  */
  
//USS
#include "../library/uss.h"
#include <hip/hip_runtime.h>

#define TESTDEBUG 1
#define BENCHMARK_MAIN 1

#if(BENCHMARK_MAIN == 1)
#include "../benchmark/dwatch.h"
#endif

//////////////////////////////////////////////
//											//
// own user-defined USS structures			//
//											//
//////////////////////////////////////////////

#define FACTORS_TO_FIND 1000
#define CANDIDATES_PER_BATCH 1000
//#define CANDIDATES_PER_BATCH_GPU 100000
#define CANDIDATES_PER_BATCH_GPU 1000
// 99488307847707803 is prime!
#define BASE_NUMBER 99488307847707802

typedef unsigned int uint;
typedef unsigned long long ullong;

#include <string>
#include <iostream>
#include <math.h>
#include <cutil.h>
#include <cutil_inline.h>

//define your personal meta_data and meta_checkpoint
struct meta_checkpoint
{
	int is_finished;
	signed long *workstatus;
	
	unsigned long long numberToTest;
	unsigned long long remainder;
	unsigned long long currentDivisor;
	unsigned long long currentSquareRoot;

	unsigned int nextIndex;
	unsigned long long primes[FACTORS_TO_FIND];
	unsigned long long exponents[FACTORS_TO_FIND];
	bool foundsolution;

	int shared_mem_available;
	hipDeviceptr_t dev_success;
	hipDeviceptr_t dev_nextIndex;
	hipDeviceptr_t dev_primes;
	hipDeviceptr_t dev_exponents;
	hipDeviceptr_t dev_currentSquareRoot;
	hipDeviceptr_t dev_remainder;
	hipStream_t cudastream;
	hipCtx_t cudaContext;
	hipFunction_t cudaFunction;
	hipModule_t cudaModule;

};
  
struct meta_data
{

};
 
//////////////////////////////////////////////
//											//
// helpers for all implementations			//
//											//
//////////////////////////////////////////////
void checkResults(struct meta_checkpoint *resources)
{
	*(resources->workstatus) = (resources->nextIndex < FACTORS_TO_FIND && resources->remainder > 1 && (resources->currentDivisor-2) <= resources->currentSquareRoot);
  if (!*(resources->workstatus))
  {
		/*****************\
		* added by Daniel *
		\*****************/
		resources->is_finished = 1;
  
		/* is remainder prime? */
		if (resources->remainder != 1 && resources->currentDivisor > resources->currentSquareRoot && resources->nextIndex < FACTORS_TO_FIND) 
		{
			resources->primes[resources->nextIndex] = resources->remainder;
			resources->exponents[resources->nextIndex] += 1;
			resources->nextIndex++;
			resources->remainder = 1;
		}
#if(TESTDEBUG == 1)		
		if (resources->nextIndex == 1) 
		{
			printf("%lld is prime!\n", resources->numberToTest);
		} 
		else 
		{
			if (resources->remainder == 1)
			{
				printf("Found ALL\n");
				for(int i = 0; i<FACTORS_TO_FIND; i++)
				{
					printf("prime: %lld multiplicity: %lld\n", resources->primes[i], resources->exponents[i]);
				}
			}	
			else
			{
				printf("Found the first\n");
			}
			printf("%i prime factors of %lld\n", resources->nextIndex, resources->numberToTest);
			if (resources->nextIndex > 0)
				printf(" (largest divisor is %lld)", resources->primes[resources->nextIndex - 1]);
			resources->foundsolution = true;
			printf("(%lld)", resources->currentDivisor);
		}
#endif

  } 
  else
	{
	*(resources->workstatus) = resources->currentDivisor;
	}
}
 

 
//////////////////////////////////////////////
//											//
// CPU implementation						//
//											//
//////////////////////////////////////////////
/* allocate cu memory and copy resources to cu */
void testCandidate(struct meta_checkpoint *resources)
{
	bool isDivisor = false;
	while ((resources->remainder % resources->currentDivisor) == 0) {
		resources->remainder /= resources->currentDivisor;
		resources->primes[resources->nextIndex] = resources->currentDivisor;
		resources->exponents[resources->nextIndex] += 1;
		isDivisor = true;
	}
	if (isDivisor) {
		resources->currentSquareRoot = ceil(sqrt((double)(resources->remainder)));
		resources->nextIndex++;
	}
}

int myalgo_cpu_init(void *md_void, void *mcp_void, int device_id)
{
	//no init for CPU
#if(TESTDEBUG == 1)
	printf("myalgo_CPU_init()\n");
#endif
	return 0;
}

int myalgo_cpu_main(void *md_void, void *mcp_void, int device_id)
{
#if(TESTDEBUG == 1)
	printf("myalgo_CPU_main()");
#endif
	//struct meta_data *md = (struct meta_data*) md_void;
	struct meta_checkpoint *resources = (struct meta_checkpoint*) mcp_void;

	*(resources->workstatus) = true;
	
	for (int candidates = 0; candidates < CANDIDATES_PER_BATCH && *(resources->workstatus); ++candidates)
	{
		testCandidate(resources);
		resources->currentDivisor += 2; //even numbers are filtered in init
		checkResults(resources);
	}

	return 0;
}

 int myalgo_cpu_free(void *md_void, void *mcp_void, int device_id)
 {
	//no cleanup for CPU
#if(TESTDEBUG == 1)	
	printf("myalgo_CPU_free()\n");
#endif
	return 0;
 }


//////////////////////////////////////////////
//											//
// CUDA	implementation						//
//											//
//////////////////////////////////////////////
/* copies the current results from the device to main memory */
static void copy_checkpoint(struct meta_checkpoint *resources)
{
	//uint *nextIndex
  cutilDrvSafeCall( hipMemcpyDtoH (&resources->nextIndex, resources->dev_nextIndex, sizeof(uint)));
	//ullong *primes
  cutilDrvSafeCall( hipMemcpyDtoH (resources->primes, resources->dev_primes, FACTORS_TO_FIND*sizeof(ullong)));
	//ullong *exponents
  cutilDrvSafeCall( hipMemcpyDtoH (resources->exponents, resources->dev_exponents, FACTORS_TO_FIND*sizeof(ullong)));
	//ullong *currentSquareRoot
  cutilDrvSafeCall( hipMemcpyDtoH (&resources->currentSquareRoot, resources->dev_currentSquareRoot, sizeof(ullong)));
	//ullong *remainder
  cutilDrvSafeCall( hipMemcpyDtoH (&resources->remainder, resources->dev_remainder, sizeof(ullong)));
}


static void execute_kernel(struct meta_checkpoint *resources)
{
  //
  // The icky part: compute the optimal number of threads per block,
  // and the number of blocks
  //
  int number_of_threads = min(CANDIDATES_PER_BATCH_GPU, 1000000); // just a guess of what is efficiently possible
	int iterations = ceil(CANDIDATES_PER_BATCH_GPU / number_of_threads);

  int threads_per_block = 512;
  int blocks_in_grid = ceil((double)number_of_threads / (double)threads_per_block);

  // set kernel parameters
	// prime_factor(ullong currentDivisor, uint number_of_threads, uint iterations, uint *succ, ullong *primes, ullong *exponents, uint *nextIndex, ullong *currentSquareRoot, ullong *remainder)
  int offset = 0;
  offset = (offset + __alignof(resources->currentDivisor) - 1) & ~(__alignof(resources->currentDivisor) - 1); // adjust offset to meet alignment requirement
  cutilDrvSafeCallNoSync(cuParamSetv(resources->cudaFunction, offset, &resources->currentDivisor, sizeof(resources->currentDivisor)));
  offset += sizeof(resources->currentDivisor);

  offset = (offset + __alignof(number_of_threads) - 1) & ~(__alignof(number_of_threads) - 1);
  cutilDrvSafeCall(cuParamSeti(resources->cudaFunction, offset, number_of_threads));
  offset += sizeof(number_of_threads);

  offset = (offset + __alignof(iterations) - 1) & ~(__alignof(iterations) - 1);
  cutilDrvSafeCall(cuParamSeti(resources->cudaFunction, offset, iterations));
  offset += sizeof(iterations);

  void* ptr = (void*)(size_t)resources->dev_success;
  offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
  cutilDrvSafeCallNoSync(cuParamSetv(resources->cudaFunction, offset, &ptr, sizeof(ptr)));
  offset += sizeof(ptr);

  ptr = (void*)(size_t)resources->dev_primes;
  offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
  cutilDrvSafeCallNoSync(cuParamSetv(resources->cudaFunction, offset, &ptr, sizeof(ptr)));
  offset += sizeof(ptr);

  ptr = (void*)(size_t)resources->dev_exponents;
  offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
  cutilDrvSafeCallNoSync(cuParamSetv(resources->cudaFunction, offset, &ptr, sizeof(ptr)));
  offset += sizeof(ptr);

  ptr = (void*)(size_t)resources->dev_nextIndex;
  offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
  cutilDrvSafeCallNoSync(cuParamSetv(resources->cudaFunction, offset, &ptr, sizeof(ptr)));
  offset += sizeof(ptr);

  ptr = (void*)(size_t)resources->dev_currentSquareRoot;
  offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
  cutilDrvSafeCallNoSync(cuParamSetv(resources->cudaFunction, offset, &ptr, sizeof(ptr)));
  offset += sizeof(ptr);

  ptr = (void*)(size_t)resources->dev_remainder;
  offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
  cutilDrvSafeCallNoSync(cuParamSetv(resources->cudaFunction, offset, &ptr, sizeof(ptr)));
  offset += sizeof(ptr);

  cutilDrvSafeCall(cuParamSetSize(resources->cudaFunction, offset));

  // set execution configuration
  dim3 block;
  block.x = threads_per_block; block.y = 1;
  cutilDrvSafeCall(cuFuncSetBlockShape(resources->cudaFunction, block.x, block.y, 1 ));

  dim3 grid;
  grid.x = blocks_in_grid; grid.y = 1;

  cutilDrvSafeCall(cuFuncSetSharedSize (resources->cudaFunction, 0));
  
  /* Launch the kernel */
  cutilDrvSafeCall(cuLaunchGridAsync( resources->cudaFunction, grid.x, grid.y, resources->cudastream ));

  /* Wait for its completion */
	cutilDrvSafeCall( hipCtxSynchronize() );
}


int myalgo_cuda_init(void *md_void, void *mcp_void, int device_id)
{
#if(TESTDEBUG == 1)
	printf("myalgo_CUDA_init()\n");
#endif
	//struct meta_data *md = (struct meta_data*) md_void;
	struct meta_checkpoint *resources = (struct meta_checkpoint*) mcp_void;
	
	/*****************\
	* added by Daniel *
	\*****************/
	/*Tobias implementation wanted a computing_unit_shortinfo
	 *containing info about available devices
	 *-> here use first device => 0
	 */
	// Initialize the driver API
	hipDevice_t cudaDevice;
	hipError_t status;
	hipInit(0);
	const char *step;

	/* Get the available number of shared memory per block */
	step = "hipDeviceGetAttribute";
	if (hipDeviceGetAttribute( &resources->shared_mem_available, hipDeviceAttributeMaxSharedMemoryPerBlock, 0 ) != hipSuccess)
	goto cleanup_and_fail;

	/* Create a context on the correct device */
	step = "hipDeviceGet";
	cutilDrvSafeCallNoSync(hipDeviceGet(&cudaDevice,0));
	step = "hipCtxCreate";
	if ((status = hipCtxCreate( &resources->cudaContext, /*Daniel: omitting param bof error hipDeviceScheduleBlockingSync | hipDeviceScheduleYield*/ 0 , cudaDevice )) != hipSuccess)
	goto cleanup_and_fail;

	/* Get the pointer to the function inside the cubin */
	step = "hipModuleLoad";
	if ((status = hipModuleLoad(&resources->cudaModule, "./testapp/prime.cubin")) != hipSuccess)
	goto cleanup_and_fail;
	step = "hipModuleGetFunction";
	if ((status = hipModuleGetFunction(&resources->cudaFunction, resources->cudaModule, "prime_factor")) != hipSuccess)
	goto cleanup_and_fail;

	/* create a stream for async operations */
	cutilDrvSafeCall( hipStreamCreateWithFlags (&resources->cudastream, 0) );

	// allocate GPU memory for match signal
	//uint *succ
	step = "Memory: dev_success";
	cutilDrvSafeCall( hipMalloc( &resources->dev_success, sizeof(uint)));
	cutilDrvSafeCall( hipMemsetD8( resources->dev_success, 0, sizeof(uint)));
	//uint *nextIndex
	step = "Memory: dev_nextIndex";
	cutilDrvSafeCall( hipMalloc( &resources->dev_nextIndex, sizeof(uint)));
	cutilDrvSafeCall( hipMemcpyHtoD (resources->dev_nextIndex, &resources->nextIndex, sizeof(uint)));
	//ullong *primes
	step = "Memory: dev_primes";
	cutilDrvSafeCall( hipMalloc( &resources->dev_primes, FACTORS_TO_FIND*sizeof(ullong)));
	cutilDrvSafeCall( hipMemcpyHtoD (resources->dev_primes, resources->primes, FACTORS_TO_FIND*sizeof(ullong)));
	//ullong *exponents
	step = "Memory: dev_exponents";
	cutilDrvSafeCall( hipMalloc( &resources->dev_exponents, FACTORS_TO_FIND*sizeof(ullong)));
	cutilDrvSafeCall( hipMemcpyHtoD (resources->dev_exponents, resources->exponents, FACTORS_TO_FIND*sizeof(ullong)));
	//ullong *currentSquareRoot
	step = "Memory: dev_currentSquareRoot";
	cutilDrvSafeCall( hipMalloc( &resources->dev_currentSquareRoot, sizeof(ullong)));
	cutilDrvSafeCall( hipMemcpyHtoD (resources->dev_currentSquareRoot, &resources->currentSquareRoot, sizeof(ullong)));
	//ullong *remainder
	step = "Memory: dev_remainder";
	cutilDrvSafeCall( hipMalloc( &resources->dev_remainder, sizeof(ullong)));
	cutilDrvSafeCall( hipMemcpyHtoD (resources->dev_remainder, &resources->remainder, sizeof(ullong)));

	return 0;

	cleanup_and_fail:
	printf("myalgo_cuda_init() failed at step ' %s ' with error ' %i ' \n", step, (int)status);
	cutilDrvSafeCall(cuCtxDetach(resources->cudaContext));	
	return 0;
}
 
int myalgo_cuda_main(void *md_void, void *mcp_void, int device_id)
{
#if(TESTDEBUG == 1)
	printf("myalgo_CUDA_main()\n");
#endif
	struct meta_checkpoint *resources = (struct meta_checkpoint*) mcp_void;

	*(resources->workstatus) = true;

	execute_kernel(resources);
	resources->currentDivisor += CANDIDATES_PER_BATCH_GPU;

	//stm << "     cuda returned after word #" << resources->currentWordNumber << endl;
	//DBG_FINE << stm.str(); stm.str("");
	uint ret = 0;
	cutilDrvSafeCall( hipMemcpyDtoH (&ret, resources->dev_success, sizeof(uint)));

	if (ret) {
	/* All factors have been found, get the data */
		copy_checkpoint(resources);
		
		checkResults(resources);
	if (*(resources->workstatus)) {
			printf("Bug: CUDA thinks all factors would have been found\n");
		}
	resources->foundsolution = true;
	}

	return 0;
}
 
int myalgo_cuda_free(void *md_void, void *mcp_void, int device_id)
{
#if(TESTDEBUG == 1)
	printf("myalgo_CUDA_free()\n");
#endif
	struct meta_checkpoint *resources = (struct meta_checkpoint*) mcp_void;

	/* copy checkpoint to main memory (only necessary if solution printing did not already) */
	if (!resources->foundsolution)
		copy_checkpoint(resources);

	/* free device resources */
	cutilDrvSafeCall(hipFree(resources->dev_success));
	cutilDrvSafeCall(hipFree(resources->dev_nextIndex));
	cutilDrvSafeCall(hipFree(resources->dev_primes));
	cutilDrvSafeCall(hipFree(resources->dev_exponents));
	cutilDrvSafeCall(hipFree(resources->dev_currentSquareRoot));
	cutilDrvSafeCall(hipFree(resources->dev_remainder));

	/* destroy execution context */
	cutilDrvSafeCall(hipModuleUnload(resources->cudaModule));
	cutilDrvSafeCall(hipStreamDestroy(resources->cudastream) );
	cutilDrvSafeCall(hipCtxDestroy(resources->cudaContext));

	return 0;
}
 
//////////////////////////////////////////////
//											//
// MAIN (fills msi and calls libuss_start)	//
//											//
//////////////////////////////////////////////
int main(int argc, char *argv[])
{
/***************************************\
* parse testapp input					*
\***************************************/
	int id = 0;
	long prime = 0;
	if(argc == 1)
	{
		//default mode: just use own BASE_NUMBER
		printf("<<< test application for uss_library>>>\n");
		prime = BASE_NUMBER;
	}
	else if(argc == 3)
	{
		id = atoi(argv[1]);
		long t = atol(argv[2]);
		if(t == 0)
		{
			//tobias mode: use id to increase base prime
			prime = BASE_NUMBER + id;
		}
		else
		{
			//manual mode: second param is user-defined prime value
			prime = t;
		}
	}
	else
	{
		printf("bad nof input parameters\n");
		exit(-1);
	}
	
	#if(BENCHMARK_MAIN == 1)
	init_dwatch();
	#endif
	
/***************************************\
* fill meta sched info (MSI) struct		*
\***************************************/
	int ret = 0;
	struct meta_sched_info msi;
	memset(&msi, 0, sizeof(struct meta_sched_info));
	
	//insert USS_ACCEL_TYPE_CPU
	ret = libuss_fill_msi(&msi, USS_ACCEL_TYPE_CPU, 2, 0, 
					&myalgo_cpu_init, 
					&myalgo_cpu_main, 
					&myalgo_cpu_free);
	if(ret == -1) {printf("Error with malloc\n"); return -1;}
	

	//insert USS_ACCEL_TYPE_CUDA
	ret = libuss_fill_msi(&msi, USS_ACCEL_TYPE_CUDA, 3, 0, 
					&myalgo_cuda_init, 
					&myalgo_cuda_main, 
					&myalgo_cuda_free);
	if(ret == -1) {printf("Error with malloc\n"); return -1;}

/***************************************\
* fill meta checkpoint (MCP)			*
\***************************************/
	struct meta_checkpoint resources;

	resources.foundsolution = false;
	resources.numberToTest = prime + 0;// * 111;  
	resources.remainder = resources.numberToTest;
	resources.currentDivisor = 3;
	resources.nextIndex = 0;
	while (resources.remainder > 1 && (resources.remainder % 2) == 0) {
		resources.remainder /= 2;
		resources.primes[0] = 2;
		resources.exponents[0] += 1;
		resources.nextIndex = 1;
	}
	resources.currentSquareRoot = ceil(sqrt((double)(resources.remainder))); 
  
	/*this variable by tobias is design to hold the status
	 *
	 *0: nothing to do any more => finished
	 *X: holds currentDivisor
	 *
	 */
	signed long workstatus = true;
	resources.workstatus = &workstatus;
	
	/*this variable is my own finished indicator
	 *0=still work to do
	 *1=finished
	 */
	resources.is_finished = 0;
	
/***************************************\
* fill meta data (MD)					*
\***************************************/
	struct meta_data md;
	//empty in this example
	
/***************************************\
* call libuss_start to enter			*
\***************************************/
	int run_on;
	int device_id;
	libuss_start(&msi, (void*)&md, (void*)&resources, &(resources.is_finished), &run_on, &device_id);
	
/***************************************\
* free meta sched info (MSI) memory		*
\***************************************/
	libuss_free_msi(&msi);
	

	#if(BENCHMARK_MAIN == 1)
	//returns id and total turnaround time in ms
	printf("%i %ld\n", id, (long)diff_dwatch());
	#endif
	return 0;
 }
