#include "hip/hip_runtime.h"
/*
 * this is a testapplication filling using the
 * library calls offered by uss_library
 */
 
/*
 * CURRENT EXAMPLE
 * 
 * increment each value of an array by one
 *
 */
  
 //USS
#include "../library/uss.h"

//basic
#include <stdlib.h>
#include <stdio.h>
//string
#include <string.h>
#include <sys/types.h>
//sleep
#include <unistd.h>
//threads
#include <pthread.h>
#define MYEXAMPLE_ARRAY 100

#define TESTDEBUG 1

//////////////////////////////////////////////
//											//
// own user-defined USS structures			//
//											//
//////////////////////////////////////////////

//define your personal meta_data and meta_checkpoint
struct meta_checkpoint
{
	int curr;
};
  
struct meta_data
{
	size_t size;
	float *host_A;
	float *host_C;
	float *dev_A;
	float *dev_C;
	int start, stop, inc_granularity, is_finished;
};
 
 
//////////////////////////////////////////////
//											//
// CPU implementation						//
//											//
//////////////////////////////////////////////
int myalgo_cpu_init(void *md_void, void *mcp_void, int device_id)
{
	//no init for CPU
	#if(TESTDEBUG == 1)	
	printf("myalgo_CPU_init()\n");
	#endif
	return 0;
}

int myalgo_cpu_main(void *md_void, void *mcp_void, int device_id)
{
	#if(TESTDEBUG == 1)	
	printf("myalgo_CPU_main()  sleeping for 1 sec  ");
	#endif
	sleep(1);
	struct meta_data *md = (struct meta_data*) md_void;
	struct meta_checkpoint *mcp = (struct meta_checkpoint*) mcp_void;
	
	int i;
	for(i = mcp->curr; i < (mcp->curr + md->inc_granularity && i < (md->stop); i++)
	{
		md->host_C[i] = md->host_A[i] + 1; 
	}
	
	mcp->curr = i; 
	#if(TESTDEBUG == 1)	
	printf("exited main with: i = %i \n", i);
	#endif
	if(i == md->stop) {md->is_finished = 1;}
	
	return 0;
}

 int myalgo_cpu_free(void *md_void, void *mcp_void, int device_id)
 {
	//no cleanup for CPU
	#if(TESTDEBUG == 1)	
	printf("myalgo_CPU_free()\n");
	#endif
	return 0;
 }


//////////////////////////////////////////////
//											//
// CUDA	implementation						//
//											//
//////////////////////////////////////////////
__global__ void VecAdd(float* A, float* C, int offset)
{
	int i = threadIdx.x;
	C[i+offset] = A[i+offset] + 2;
}

int myalgo_cuda_init(void *md_void, void *mcp_void, int device_id)
{
	#if(TESTDEBUG == 1)	
	printf("myalgo_CUDA_init()\n");
	#endif
	sleep(1);
	struct meta_data *md = (struct meta_data*) md_void;
	hipError_t retruntime;
	
	retruntime = hipSetDevice(0);	
	if (retruntime != hipSuccess) {printf("SetDevice Error%i\n", (int)retruntime); exit(1);}
	
	//get space on cuda
	retruntime = hipMalloc(&(md->dev_A), md->size);
	if (retruntime != hipSuccess) {printf("Error malloc: %i\n", (int)retruntime); exit(1);}
	retruntime = hipMalloc(&(md->dev_C), md->size);
	if (retruntime != hipSuccess) {printf("Error malloc %i\n", (int)retruntime); exit(1);}
	
	//copy original data and modified vector completely
	retruntime = hipMemcpy(md->dev_A, md->host_A, md->size, hipMemcpyHostToDevice);
	if (retruntime != hipSuccess) {printf("Error Memcopy %i\n", (int)retruntime); exit(1);}
	retruntime = hipMemcpy(md->dev_C, md->host_C, md->size, hipMemcpyHostToDevice);
	if (retruntime != hipSuccess) {printf("Error Memcopy %i\n", (int)retruntime); exit(1);}
	
	return 0;
}
 
int myalgo_cuda_main(void *md_void, void *mcp_void, int device_id)
{
	#if(TESTDEBUG == 1)	
	printf("myalgo_CUDA_main()  sleeping for 1 sec  ");
	#endif
	sleep(1);
	struct meta_data *md = (struct meta_data*) md_void;
	struct meta_checkpoint *mcp = (struct meta_checkpoint*) mcp_void;
	//
	//run kernel	
	//
	int threadsPerBlock, blocksPerGrid, N;
	//run for md->inc_granularityor less if we are close to stop
	if(mcp->curr + md->inc_granularity< md->stop)
	{
		N = md->inc_granularity;
	}
	else
	{
		N = md->stop - mcp->curr + 1;
	}
	
	
	threadsPerBlock = 256;
	blocksPerGrid = (N + threadsPerBlock -1) / threadsPerBlock;
	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(md->dev_A, md->dev_C, mcp->curr);

	if(mcp->curr + md->inc_granularity< md->stop)
	{
		mcp->curr += (N); 
	}
	else
	{
		mcp->curr += (N-1);
	}
	
	#if(TESTDEBUG == 1)	
	printf("exited main with: mcp->curr = %i \n", mcp->curr);
	#endif
	
	if(mcp->curr == md->stop) {md->is_finished = 1;}
	if(mcp->curr >  md->stop) {printf("Error reached value greater than md->stop\n"); exit(-1);}
	
	return 0;
}
 
int myalgo_cuda_free(void *md_void, void *mcp_void, int device_id)
{
	#if(TESTDEBUG == 1)	
	printf("myalgo_CUDA_free()\n");
	#endif
	struct meta_data *md = (struct meta_data*) md_void;
	hipError_t retruntime;
	
	//copy result back
	retruntime = hipMemcpy(md->host_C, md->dev_C, md->size, hipMemcpyDeviceToHost);
	if (retruntime != hipSuccess) {printf("Error malloc \n"); exit(1);}
	
	//free device memory
	if(md->dev_C) hipFree(md->dev_C);

	return 0;
}
 
//////////////////////////////////////////////
//											//
// MAIN (fills msi and calls libuss_start)	//
//											//
//////////////////////////////////////////////
int main(int argc, char *argv[])
 {
/***************************************\
* parse testapp input					*
\***************************************/
	int i, id = 0;
	int inc_granularity= 10;
	if(argc == 1)
	{
		//default mode
		printf("<<< test application for uss_library>>>\n");
		inc_granularity= 10;
	}
	else if(argc == 3)
	{
		//benchmark mode
		id = atoi(argv[1]);
		inc_granularity= atoi(argv[2]);
	}
	else
	{
		printf("bad nof input parameters\n");
		exit(-1);
	}
	
/***************************************\
* fill meta sched info (MSI) struct		*
\***************************************/
	int ret = 0;
	struct meta_sched_info msi;
	memset(&msi, 0, sizeof(struct meta_sched_info));
	
	//insert USS_ACCEL_TYPE_CPU
	ret = libuss_fill_msi(&msi, USS_ACCEL_TYPE_CPU, 2, 0, 
					&myalgo_cpu_init, 
					&myalgo_cpu_main, 
					&myalgo_cpu_free);
	if(ret == -1) {printf("Error with malloc\n"); return -1;}
	

	//insert USS_ACCEL_TYPE_CUDA
		ret = libuss_fill_msi(&msi, USS_ACCEL_TYPE_CUDA, 3, 0, 
					&myalgo_cuda_init, 
					&myalgo_cuda_main, 
					&myalgo_cuda_free);
	if(ret == -1) {printf("Error with malloc\n"); return -1;}

/***************************************\
* fill meta checkpoint (MCP)			*
\***************************************/
	struct meta_checkpoint mcp;
	mcp.curr = 0;
	
/***************************************\
* fill meta data (MD)					*
\***************************************/
	struct meta_data md;
	
	md.host_A = (float*) malloc(MYEXAMPLE_ARRAY*sizeof(float)); if(!(md.host_A)) {printf("Error with malloc\n"); exit(0);}
	memset(md.host_A, 0, MYEXAMPLE_ARRAY*sizeof(float));
	md.host_C = (float*) malloc(MYEXAMPLE_ARRAY*sizeof(float)); if(!(md.host_C)) {printf("Error with malloc\n"); exit(0);}
	memset(md.host_A, 0, MYEXAMPLE_ARRAY*sizeof(float));
	
	md.size = sizeof(float)*MYEXAMPLE_ARRAY;
	md.stop = MYEXAMPLE_ARRAY - 1;
	md.inc_granularity= inc_granularity;
	md.is_finished = 0;
	
/***************************************\
* call libuss_start to enter			*
\***************************************/
	#if(TESTDEBUG == 1)
	printf("test data before algo:\n");
	for(i = 0; i < md.stop; i++) {printf("%i ", (int)md.host_C[i]);}
	#endif
	
	int run_on;
	int device_id;
	libuss_start(&msi, (void*)&md, (void*)&mcp, &(md.is_finished), &run_on, &device_id);
	
	#if(TESTDEBUG == 1)
	printf("test data after algo:\n");
	for(i = 0; i < md.stop; i++) {printf("%i ", (int)md.host_C[i]);}
	#endif
	
/***************************************\
* free meta sched info (MSI) memory		*
\***************************************/
	libuss_free_msi(&msi);
	
	return 0;
 }
